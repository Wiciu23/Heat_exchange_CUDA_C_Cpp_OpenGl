#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <GL/glew.h>
#include <GLFW/glfw3.h>
#include <cstdlib>
#include <stdlib.h>
#define WIDTH 512
#define HEIGHT 512
#define COUNT WIDTH*HEIGHT
#define MAX_TEMP 100

__global__ void kernel(double *temperatura,double *temperatura_bufor)
{
	//__shared__ double blok[16][16];

  int x = threadIdx.x + blockIdx.x * blockDim.x;
  int y = threadIdx.y + blockIdx.y * blockDim.y;

  int offset = x + y * blockDim.x * gridDim.x;

   
	  if(x != 0 && x != gridDim.x*blockDim.x-1 && y != 0 && y != gridDim.y*blockDim.y-1){

		  temperatura_bufor[offset] = (temperatura[offset-1]+temperatura[offset+1]+temperatura[offset+blockDim.x*gridDim.x]+temperatura[offset-blockDim.x*gridDim.x])/4;
	  }else
	  {
		  temperatura_bufor[offset] = temperatura[offset];
	  }
		  __syncthreads();	
}

__global__ void colorToBitmap(float * pixels, double *temperatura)
  {
  
   
  int x = threadIdx.x + blockIdx.x * blockDim.x;
  int y = threadIdx.y + blockIdx.y * blockDim.y;

  int offset = x + y * blockDim.x * gridDim.x;

  float kolor = (float)(temperatura[offset]/MAX_TEMP);

	pixels[offset*3] = 0;
	pixels[offset*3+1] = kolor;
	pixels[offset*3+2] = 0 ;

  /*
  if(kolor > 0.7){
	  pixels[offset*3] = kolor;
	  pixels[offset*3+1] = 0;
	  pixels[offset*3+2] = 0 ;
  }else if(kolor < 0.7 && kolor > 0.4)
  {
	  pixels[offset*3] = kolor;
	  pixels[offset*3+1] = kolor;
	  pixels[offset*3+2] = 0;
  }else
  {
	  pixels[offset*3] = 0;
	  pixels[offset*3+1] = kolor;
	  pixels[offset*3+2] = 0.4-kolor ;
  }

  */
  

}

  void swap(double **bufor1, double **bufor2){
	  double *temp;
	  temp  = *bufor1;
	  *bufor2 = *bufor1;
	  *bufor1 = temp;
  }


void RenderScene(float * pixels);

int main()
{
	float * dst =  (float *)calloc(HEIGHT*WIDTH*3,sizeof(float));

	double *temperatura_h = (double *)calloc(HEIGHT*HEIGHT,sizeof(double));

	double *temperatura_d;
	hipMalloc(&temperatura_d,HEIGHT*HEIGHT*sizeof(double));

	//warunki poczatkowe 100 st. na lewej scianie.
	for (int i = 0; i < WIDTH; i++)
	{ 
		for (int j = 0; j < HEIGHT; j++)
		{		
			if(j == 0)
			{
				int index = i*WIDTH+j;
				temperatura_h[index]=MAX_TEMP;
			}
		}
	}

	double *temperatura_bufor_d;
	hipMalloc(&temperatura_bufor_d,HEIGHT*HEIGHT*sizeof(double));

	hipMemcpy(temperatura_d,temperatura_h,WIDTH*HEIGHT*sizeof(double),hipMemcpyHostToDevice);
	hipMemcpy(temperatura_bufor_d,temperatura_h,WIDTH*HEIGHT*sizeof(double),hipMemcpyHostToDevice);
	int size = WIDTH*HEIGHT*sizeof(float)*3; // BITMAPA RGB


  float* src;
  hipMalloc(&src, size);
  
  dim3 blocks(WIDTH/16, HEIGHT/16);
  dim3 threads(16,16);
  for (int i = 0; i < 50000; i++)
  {
	kernel<<<blocks,threads>>>(temperatura_d,temperatura_bufor_d);
	swap(&temperatura_bufor_d,&temperatura_d);
  }
  
  colorToBitmap<<<blocks,threads>>>(src,temperatura_d);
  hipMemcpy(dst, src, size, hipMemcpyDeviceToHost);
  
  glfwInit();

    GLFWwindow* window = glfwCreateWindow(WIDTH, HEIGHT, "CudaC/C++ Boltzman Method", nullptr, nullptr);    
  
    glfwMakeContextCurrent(window);

    while (!glfwWindowShouldClose(window)){
     
		glClear(GL_COLOR_BUFFER_BIT);

		RenderScene(dst);

		glFlush();

		glfwSwapBuffers(window);

		glfwWaitEvents();
    }

  hipFree(src);
  hipFree(temperatura_bufor_d);
  hipFree(temperatura_d);
  return 0;
}


void RenderScene(float * pixels){
	glDrawPixels(WIDTH, HEIGHT, GL_RGB, GL_FLOAT, pixels);
}



